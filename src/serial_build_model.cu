#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/unique.h>
#include <thrust/count.h>
#include <thrust/iterator/counting_iterator.h>
#include "cnpy.h"
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <chrono>
using namespace std::chrono;
#include <iostream>

long long int ncells;
long long int NcNr;    
long long int NcNa;    
long long int NcNrNa;   


/*
------  Declarations of utility functions from utils.h -------
*/
cnpy::NpyArray read_velocity_field_data( std::string file_path_name, int* n_elements);
void define_xs_or_ys(float* xs, float dx, float x0, int gsize);
void save_master_Coos_to_file(std::string op_FnamePfx, int num_actions, 
thrust::host_vector<long long int> &H_master_cooS1, 
    thrust::host_vector<long long int> &H_master_cooS2, 
    thrust::host_vector<float> &H_master_cooVal,
    thrust::host_vector<float> &H_master_R,
    thrust::host_vector<long long int>* H_Aarr_of_cooS1,
    thrust::host_vector<long long int>* H_Aarr_of_cooS2,
    thrust::host_vector<float>* H_Aarr_of_cooProb,
    thrust::host_vector<float>* H_Aarr_of_Rs,
    thrust::host_vector<float> &prob_params,
    long long int* DP_relv_params,
    unsigned long int num_DP_params);


// template<typename dType> template not working for thrust vectors
void make_dir(std::string dir_name);
void populate_ac_angles(float* ac_angles, int num_ac_angles);
void populate_ac_speeds(float* ac_speeds, int num_ac_speeds, float Fmax);
void populate_actions(float** H_actions, int num_ac_speeds, int num_ac_angles, float Fmax);



long long int state1D_from_spid(int32_t T, int32_t sp_id, long long int ncells){   
    // j ~ blockIdx.x
    // i ~ blockIdx.y 
    // The above three consitute a spatial state index from i and j of grid
    // last term is for including time index as well.

    // return value for chunks concept
    return sp_id + (T*ncells);
}


long long int state1D_from_ij(int32_t*  posid, int32_t T, int32_t gsize){
    // posid = {i , j}
    // state id = j + i*dim(i) + T*dim(i)*dim(j)

    // return value for chunks concept
    return (posid[1] + posid[0]*gsize + (T*gsize*gsize)*1LL ) ; 

}

//TODO
// int32_t get_rzn_id(){
//     return (blockIdx.z * blockDim.x)  + threadIdx.x;
// }


void get_posids_from_sp_id(long long int sp_id, int gsize, int32_t* posids){
    posids[0] = sp_id/gsize;
    posids[1] = sp_id%gsize;
    return;
}


long long int get_sp_id(int i , int j, int gsize){
    // sp_id: 1d spatial id ranging from 0 to ncells
    long long int sp_id = j + (i*gsize)*1LL;
    return sp_id;
}


void get_posids_relS2_0(int32_t m, int32_t* posids_S1, int32_t* posids_relS2_0){
    // m*m is size of neighbour grid
    // returns i,j index of upper left corner of neighbour grid
    int32_t i1 = posids_S1[0];
    int32_t j1 = posids_S1[1];   
    int32_t del = (m-1)/2;
    posids_relS2_0[0] =  i1 - del;
    posids_relS2_0[1] =  j1 - del;
    return;
}


long long int get_rel_sp_id2(int32_t m, int32_t* posids_S2, int32_t* posids_relS2_0){
    // returns relative sp_id for S2 in neighbour grid

    int32_t del_i = posids_S2[0] - posids_relS2_0[0]; // i2 - rel_i0
    int32_t del_j = posids_S2[1] - posids_relS2_0[1]; // j2 - rel_j0
    long long int rel_sp_id2;
    // if S2 outstde neighbor grid, assign special value to rel_sp_id2
    // this will help keep count of no. of S2s that lie outside neighb grid
    // ideally there should be 0 such S2s
    if (del_i < m && del_j < m)
        rel_sp_id2 = del_j + (m*del_i);
    else 
        rel_sp_id2 = m*m;               

    return rel_sp_id2;
}



long long int get_sp_id2_from_rel_sp_id2(int32_t m, int32_t gsize, 
                                    long long int rel_sp_id2, int32_t* posids_relS2_0){
    // returns Sp_id2 from rel_sp_id2
    long long int sp_id2;
    int32_t del_i = rel_sp_id2/m;
    int32_t del_j = rel_sp_id2%m;

    int32_t i2 = del_i + posids_relS2_0[0];
    int32_t j2 = del_j + posids_relS2_0[1];

    sp_id2 = j2 + gsize*i2;

    return sp_id2;
}



bool is_edge_state(int32_t i, int32_t j, int gsize){
    // n = gsize -1 that is the last index of the domain assuming square domain
    int32_t n = gsize - 1;
    if (i == 0 || i == n || j == 0 || j == n ) 
        return true;
    else 
        return false;
}


bool is_in_obstacle(long long int sp_id, int T, long long int ncells, int* all_mask_mat){
    //returns true if obstacle is present in state T,i,j

    long long int mean_id = state1D_from_spid(T, sp_id, ncells);
    // std::cout<< "--check 8 : mean_id = " << mean_id <<  "\n";
    // std::cout << "T = " << T << "\n";
    // std::cout << "sp_id = " << sp_id << "\n";
    // std::cout << "ncells = " << ncells << "\n";


    return (all_mask_mat[mean_id] == 1 );

}


bool is_terminal(int32_t i, int32_t j, float* params){
    // terminal state indices (of UL corner of terminal subgrid if term_subgrid_size>1)
    int32_t i_term = params[8];         
    int32_t j_term = params[9];
    int tsgsize = params[12]; //term_subgrid_size

    if( (i >= i_term && i < i_term + tsgsize)  && (j >= j_term && j < j_term + tsgsize) )
        return true;
    else return false;
}


bool my_isnan(int s){
    // By IEEE 754 rule, NaN is not equal to NaN
    return s != s;
}


void get_xypos_from_ij(int32_t i, int32_t j, int32_t gsize ,float* xs, float* ys, float* x, float* y){
    *x = xs[j];
        // *y = ys[gridDim.x - 1 - i];
    *y = ys[gsize - 1 - i];

    return;
}


long long int get_sp_id_from_posid(int32_t* posids, int32_t gsize){
    // gives sp_id from posids (i,j)
    return posids[1] + (gsize*posids[0]*1LL) ;
}


float get_angle_in_0_2pi(float theta){
    float f_pi = 3.141592;
    if (theta < 0)
        return theta + (2*f_pi);
    else
        return theta;
}


float calculate_one_step_reward(float ac_speed, float ac_angle, float rad1, float rad2, float* params){

    int method = params[13];
    float Cr = 1;       // coeffecient for radaition term
    float Cf = 1;       // coeffecient for energy consumtion
    float Ct = 0.01;   // small coeffecient for time for to prevent lazy start
    float dt = params[4];

    if (method == 0)    //time
        return -dt;

    else if (method == 1){   //energy1
        return -(Cf*ac_speed*ac_speed + Ct)*dt;
    } 

    else if (method == 2){  //energy2: maximise (collection-consumption)
        return ((Cr*(rad2 + rad1)/2) - (Cf*ac_speed*ac_speed) - Ct)*dt;
    }

    else
        return 0;

}


void move(float ac_speed, float ac_angle, float vx, float vy, int32_t T, float* xs, float* ys, int32_t* posids, float* params, float* r ){
    int32_t gsize = params[0];
    int32_t n = params[0] - 1;      // gsize - 1
    // int32_t num_actions = params[1];
    // int32_t nrzns = params[2];
    // float F = params[3];
    int32_t nt = params[10];
    float F = ac_speed;
    float dt = params[4];
    float r_outbound = params[5];
    float r_terminal = params[6];
    // int32_t nT = params[10];
    float Dj = fabsf(xs[1] - xs[0]);
    float Di = fabsf(ys[1] - ys[0]);
    int32_t i0 = posids[0];
    int32_t j0 = posids[1];
    // std::cout << "posids = " << posids[0] << ", " << posids[1] << "\n";
    // std::cout << "i0, j0 = " << i0 << ", " << j0 << "\n";

    float vnetx = F*cosf(ac_angle) + vx;
    float vnety = F*sinf(ac_angle) + vy;
    float x, y;
    get_xypos_from_ij(i0, j0, gsize, xs, ys, &x, &y); // x, y stores centre coords of state i0,j0
    float xnew = x + (vnetx * dt);
    float ynew = y + (vnety * dt);
    // std::cout << "x, y = " << x << ", " << y << "\n";
    // std::cout << "vx, vy = " << vx << ", " << vy << "\n";
    // std::cout << "vnetx, vnety = " << vnetx << ", " << vnety << "\n";
    // std::cout << "xnew, ynew = " << xnew << ", " << ynew << "\n";

    // float r_step = 0;
    *r = 0;         // intiilaise r with 0

    if (xnew > xs[n])
        {
            xnew = xs[n];
            *r += r_outbound;
        }
    else if (xnew < xs[0])
        {
            xnew = xs[0];
            *r += r_outbound;
        }
    if (ynew > ys[n])
        {
            ynew =  ys[n];
            *r += r_outbound;
        }
    else if (ynew < ys[0])
        {
            ynew =  ys[0];
            *r += r_outbound;
        }
    // TODO:xxDONE check logic wrt remainderf. remquof had issue
    int32_t xind, yind;
    //float remx = remquof((xnew - xs[0]), Dj, &xind);
    //float remy = remquof(-(ynew - ys[n]), Di, &yind);
    float remx = remainderf((xnew - xs[0]), Dj);
    float remy = remainderf(-(ynew - ys[n]), Di);
    xind = ((xnew - xs[0]) - remx)/Dj;
    yind = (-(ynew - ys[n]) - remy)/Di;
    if ((remx >= 0.5 * Dj) && (remy >= 0.5 * Di))
        {
            xind += 1;
            yind += 1;
        }
    else if ((remx >= 0.5 * Dj && remy < 0.5 * Di))
        {
            xind += 1;
        }
    else if ((remx < 0.5 * Dj && remy >= 0.5 * Di))
        {
            yind += 1;
        }
    if (!(my_isnan(xind) || my_isnan(yind)))
        {   
            // update posids
            posids[0] = yind;
            posids[1] = xind;
            if (is_edge_state(posids[0], posids[1], gsize))  //line 110
                {
                    *r += r_outbound;
                }
            
        }

    // r_step = calculate_one_step_reward(ac_speed, ac_angle, xs, ys, i0, j0, x, y, posids, params, vnetx, vnety);
    // // r_step = -dt;
    // *r += r_step; //TODO: numerical check remaining
    if (is_terminal(posids[0], posids[1], params)){
            *r += r_terminal;
    }
    else{
            //reaching any state in the last timestep which is not terminal is penalised
            if (T == nt-2)
                *r += r_outbound; 
    }

    return;
}



void extract_velocity(int32_t rzn_id, int32_t* posids, long long int sp_id, long long int ncells, float* vx, float* vy,
    int32_t T, float* all_u_mat, float* all_v_mat, float* all_ui_mat, 
    float* all_vi_mat, float* all_Yi, float* params){

    int32_t nrzns = params[2];
    int32_t nmodes = params[7];    
    int32_t gsize = params[0];          

    long long int sp_uvi, str_uvi, sp_Yi; //startpoints and strides for accessing all_ui_mat, all_vi_mat and all_Yi
    // int str_Yi;
    float sum_x = 0;
    float sum_y = 0;
    float vx_mean, vy_mean;
    //rzn_id: rzn index to identify which of the 5k rzn it is. used to access all_Yi.

    //mean_id is the index used to access the flattened all_u_mat[t,i,j].
    long long int mean_id = state1D_from_spid(T, sp_id, ncells);
    //to access all_ui_mat and all_vi_mat

    str_uvi = gsize*gsize*1LL;
    sp_uvi = (T * nmodes * str_uvi) + (gsize * posids[0]) + (posids[1]);

    // to access all_Yi
    sp_Yi = (T * nrzns * nmodes * 1LL) + (rzn_id * nmodes);
    vx_mean = all_u_mat[mean_id];
    for(int i = 0; i < nmodes; i++)
    {
    sum_x += all_ui_mat[sp_uvi + (i*str_uvi)]*all_Yi[sp_Yi + i];
    }
    vy_mean = all_v_mat[mean_id];
    for(int i = 0; i < nmodes; i++)
    {
    sum_y += all_vi_mat[sp_uvi + (i*str_uvi)]*all_Yi[sp_Yi + i];
    }

    *vx = vx_mean + sum_x;
    *vy = vy_mean + sum_y;

    return;
}


void extract_radiation(long long int sp_id, int32_t T, long long int ncells, 
                                float* all_s_mat, float* rad){
    // for DETERMINISTIC radiation (scalar) field
    // extract radiation (scalar) from scalar matrix 
    
    long long int mean_id = state1D_from_spid(T, sp_id, ncells);
    *rad = all_s_mat[mean_id];

    return;
}


bool is_within_band(int i, int j, int i1, int j1, int i2, int j2, float* xs, float* ys, int gsize){
    //returns true if i,j are within the band connecticng cells i1,j1 and i2,j2
    if(i1==i2 || j1==j2){
        return true;
    }
    else{
        float x, y, x1, y1, x2, y2;
        float cell_diag = fabsf(xs[1]-xs[0])*1.414213;
        get_xypos_from_ij(i, j, gsize, xs, ys, &x, &y); // x, y stores centre coords of state i0,j0
        get_xypos_from_ij(i1, j1, gsize, xs, ys, &x1, &y1); 
        get_xypos_from_ij(i2, j2, gsize, xs, ys, &x2, &y2);
        float A = (y2-y1)/(x2-x1);
        float B = -1;
        float C = y1 - (A*x1);
        float dist_btw_pt_line = fabsf(A*x + B*y + C)/sqrtf((A*A) + (B*B));
        
        if (dist_btw_pt_line < cell_diag)
            return true;
        else
            return false;
    }
}


bool goes_through_obstacle(long long int sp_id1, long long int sp_id2, int T, 
                                        long long int ncells, int* D_all_mask_mat, 
                                        float* xs, float* ys, float* params){

    // returns true if the transition involves going through obstacle
    // std::cout<< "inside func goes_through_obstacle\n";
    // std::cout<< "--check pre-5 \n";
    // std::cout << "sp_id1 = " << sp_id1 << "\n";
    // std::cout << "sp_id2 = " << sp_id2 << "\n";

    bool possible_collision = false;
    int posid1[2];
    int posid2[2];
    int gsize = params[0];
    long long int sp_id;
    get_posids_from_sp_id(sp_id1, gsize, posid1);
    get_posids_from_sp_id(sp_id2, gsize, posid2);
    int imin = min(posid1[0], posid2[0]);
    int imax = max(posid1[0], posid2[0]);
    int jmin = min(posid1[1], posid2[1]);
    int jmax = max(posid1[1], posid2[1]);
    // std::cout<< "--check 5 \n";
    // std::cout << "imin = " << imin << "\n";
    // std::cout << "imax = " << imax << "\n";
    // std::cout << "jmin = " << jmin << "\n";
    // std::cout << "jmax = " << jmax << "\n";
    

    for(int i=imin; i<=imax; i++){
        for(int j=jmin; j<=jmax; j++){
            if(!(i==posid1[0]&&j==posid1[1])){
                sp_id = j + gsize*i*1LL ;
                // std::cout << "check 5a: sp_id = " << sp_id << "\n";
                // std::cout << "gsize = " << gsize << "\n";
                // std::cout << "i = " << i << "\n";

                if ( is_in_obstacle(sp_id, T, ncells, D_all_mask_mat) || is_in_obstacle(sp_id, T+1, ncells, D_all_mask_mat)){
                    // std::cout<< "--check 6 \n";
                    if (is_within_band(i, j, posid1[0], posid1[1], posid2[0], posid2[1], xs, ys, gsize) == true){
                        // std::cout<< "--check 7 \n";
                        possible_collision = true;
                        return true;
                    }
                }
            }
        }
    }
    
    return possible_collision;
}




//TODO
    // transition_calc(D_T_arr, 
    //     ncells, all_u_arr, all_v_arr, all_ui_arr, all_vi_arr, all_yi_arr,
    //     all_s_arr, all_mask_arr,
    //     ac_speed, ac_angle, xs, ys, params, H_master_sumRsa_arr, 
    //     H_master_S2_arr);
void transition_calc(float* T_arr, long long int ncells, 
                            float* all_u_mat, float* all_v_mat, float* all_ui_mat, float* all_vi_mat, float* all_Yi,
                            float* D_all_s_mat, int* D_all_mask_mat,
                            float ac_speed, float ac_angle, float* xs, float* ys, float* params, float* sumR_sa, 
                            float* results){
                            // resutls directions- 1: along S2;  2: along S1;    3: along columns towards count
    
    
    
    int32_t gsize = params[0];          // size of grid along 1 direction. ASSUMING square grid.
    int32_t nrzns = params[2]; 
    float r_outbound = params[5];        
    // int32_t is_stationary = params[11];
    int32_t T = (int32_t)T_arr[0];      // current timestep
    int32_t idx;
    long long int res_idx;
    float vx, vy, rad1, rad2;
    int32_t rzn_id;
    long long int sp_id;      //sp_id is space_id. S1%(gsize*gsize)
    long long int sp_id2;
    long long int rel_sp_id2;
    int32_t posids_relS2_0[2];
    int32_t posids_S1[2];   //doesn't get updated
    int32_t posids[2];  //get usdated  //static declaration of array of size 2 to hold i and j values of S1. 
    int32_t m = (int32_t) params[18];
    int32_t Nb = (m*m) + 1;
    float one = 1.0;
    
    for(int i=0; i<gsize; i++){

        for(int j=0; j<gsize; j++){

            for(int k=0; k<nrzns; k++){
                // std::cout << "i,j,k" << i << ", " << j << ", " << k << "\n";
                // idx = get_thread_idx();
                sp_id = get_sp_id(i,j,gsize);      //sp_id is space_id. S1%(gsize*gsize)

                // std::cout<< "get posids from spid\n";
                get_posids_from_sp_id(sp_id, gsize, posids);
                get_posids_from_sp_id(sp_id, gsize, posids_S1);
                rzn_id = k;
                // std::cout << "posids = " << posids[0] << ", " << posids[1] << "\n";
                // std::cout << "posids__S1 = " << posids_S1[0] << ", " << posids_S1[1] << "\n";

                //  Afer move() these will be overwritten by i and j values of S2
                float r=0;              // to store immediate reward
                float r_step;

                // std::cout<< "extract velocity and radiation\n";
                
                extract_velocity(rzn_id, posids, sp_id, ncells, &vx, &vy, T, all_u_mat, all_v_mat, all_ui_mat, all_vi_mat, all_Yi, params);
                extract_radiation(sp_id, T, ncells, D_all_s_mat, &rad1);
                // std::cout<< "check for obstacles\n";

                // if s1 not terminal
                if (is_terminal(posids[0], posids[1], params) == false){
                    // if s1 not in obstacle
                    // std::cout<< "--check 1\n";

                    if (is_in_obstacle(sp_id, T, ncells, D_all_mask_mat) == false){
                        // std::cout<< "--check 2\n";

                        // moves agent and adds r_outbound and r_terminal to r
                        move(ac_speed, ac_angle, vx, vy, T, xs, ys, posids, params, &r);
                        // std::cout << "posids = " << posids[0] << ", " << posids[1] << "\n";
                        // std::cout << "posids__S1 = " << posids_S1[0] << ", " << posids_S1[1] << "\n";
        
                        sp_id2 = get_sp_id_from_posid(posids, gsize);
                        // std::cout << "sp_id2 = " << sp_id2 << "\n";


                        // extract_radiation(sp_id2, T+1, ncells, D_all_s_mat, &rad2);
                        rad2=0;
                        // std::cout<< "--check 3 post move\n";

                        // adds one step-reward based on method. mehthod is available in params
                        r_step = calculate_one_step_reward(ac_speed, ac_angle, rad1, rad2, params);
                        r += r_step;
                        // std::cout<< "--check 4 post onestep\n";

                        // if S2 is an obstacle cell. then penalise with r_outbound
                        // if (is_in_obstacle(sp_id2, T+1, ncells, D_all_mask_mat) == true )
                        //     r = r_outbound;
                        if (goes_through_obstacle(sp_id, sp_id2, T, ncells, D_all_mask_mat, xs, ys, params) == true)
                            r = r_outbound;
                        
                        // std::cout<< "--check 5 post goesthouthg\n";

                    }
                    // if s1 is in obstacle, then no update to posid
                    else
                        r = r_outbound;
                }
                // std::cout<< "get_posids_relS2_0\n";

                get_posids_relS2_0(m, posids_S1, posids_relS2_0);
                rel_sp_id2 = get_rel_sp_id2(m, posids, posids_relS2_0);
                res_idx = sp_id*Nb + rel_sp_id2;
                results[res_idx] += 1;
                //writing to sumR_sa. this array will later be divided by nrzns, to get the avg
                sumR_sa[sp_id]+=r;
                // float a = atomicAdd(&sumR_sa[sp_id], r); 

            }
        }
    }

    return;
}


//TODO
void compute_mean(float* D_master_sumRsa_arr, int size, int nrzns) {
    // computes mean
    for(int i=0; i<size; i++){
        D_master_sumRsa_arr[i] =  D_master_sumRsa_arr[i]/nrzns;
    }
    return;
}


//TODO
void count_kernel(float* D_master_S2_arr_ip,long long int ncells, int Nb, int nrzns, unsigned long long int* num_uq_s2_ptr) {
    // D_master_S2_arr_ip contains count of relS2s for S1s for a given action
    // This kernel counts no. of nnz elements for a given S1
    // This is needed for getting total nnz to initiliase COO matrix
    // ncells is gridDim,  i.e. we have ncells blocks in grid
    // Nb is blockDim, i.e we have Nb threads in block
    int idx;
    for(int i=0; i<ncells; i++){
        int count = 0;
        for(int j=0; j<Nb-1; j++){
            idx = Nb*i + j;
            if (D_master_S2_arr_ip[idx]!=0){
                count += 1;
            }
        num_uq_s2_ptr[i] = count;

        }
    }

    return;
}

//TODO
void reduce_kernel(float* D_master_S2_arr_ip, int t, int Nb, int m,
                            long long int ncells, int nrzns, int gsize, 
                            long long int* D_coo_s1_arr, long long int* D_coo_s2_arr, 
                            float* D_coo_cnt_arr, unsigned long long int* num_uq_s2_ptr, unsigned long long int* prSum_num_uq_s2_ptr){

    // long long int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
    long long int start_idx; // to access tid'th threads 0-pos in ip_arr

    long long int n_uqs; //number of unique S2s for tid'th block
    long long int op_st_id ;   //sum of number of uniqeu S2s uptil tid'th block. to access tid'th thread's 0-pos in op_arr

    long long int ith_nuq; //ranges from 0 to n_uqs , to index number between 0 and n_uqs
    long long int s1;
    long long int rel_sp_id2;
    long long int sp_id2;
    long long int S2;
    long long int sp_id1;
    float count; //first if eval will lead to else condition and do  count++ 
    int32_t posids_relS2_0[2];
    int32_t posids_S1[2];


    for(int k=0; k<ncells; k++){
        ith_nuq = 0; //ranges from 0 to n_uqs , to index number between 0 and n_uqs
        start_idx = k*Nb; // to access tid'th threads 0-pos in ip_arr
        n_uqs = num_uq_s2_ptr[k]; //number of unique S2s for tid'th block or the kth cell
        op_st_id = prSum_num_uq_s2_ptr[k];
        sp_id1 = k;
        s1 = k + (t*ncells);
        for(long long int i = 0; i< n_uqs; i++)
            D_coo_s1_arr[op_st_id + i] = s1;
        
        for(long long int i = 0; i< Nb-1; i++){
            count = D_master_S2_arr_ip[start_idx + i];
            if (count != 0){
                rel_sp_id2 = i;
                get_posids_from_sp_id(sp_id1, gsize, posids_S1);
                get_posids_relS2_0(m, posids_S1, posids_relS2_0);
                sp_id2 = get_sp_id2_from_rel_sp_id2(m, gsize, 
                    rel_sp_id2, posids_relS2_0);
                S2 = state1D_from_spid(t, sp_id2, ncells);
                D_coo_s2_arr[op_st_id + ith_nuq] = S2;         // store old_s2 value in the [.. + ith] position
                D_coo_cnt_arr[op_st_id + ith_nuq] = count/nrzns;   // store prob value in the [.. + ith] position
                ith_nuq++;                                      // increment i
            }
        }
    }

   return;
}


template<typename dType>
void print_array(dType* array, int num_elems,std::string array_name, std::string end){
    std::cout << array_name << std::endl;
    for(int i = 0; i < num_elems; i++)
        std::cout << array[i] << " " << end;
    std::cout << std::endl;
}


std::string get_prob_name(int num_ac_speeds, int num_ac_angles, int i_term, int j_term,
                            int tsg_size){

    std::string s_n_ac_sp = std::to_string(num_ac_speeds);
    std::string s_n_ac_ac = std::to_string(num_ac_angles);
    std::string s_i = std::to_string(i_term);
    std::string s_j = std::to_string(j_term);
    std::string s_tsg = std::to_string(tsg_size);

    std::string name = "a" + s_n_ac_sp + "x" + s_n_ac_ac + "_" 
                        + "i" + s_i + "_" "j" + s_j + "_"
                        + "ref" + s_tsg;

    return name;
}









void build_sparse_transition_model_at_T_at_a(int t, int action_id, int bDimx, thrust::host_vector<float> &H_tdummy, 
                                float* all_u_arr, float* all_v_arr, float* all_ui_arr,
                                float* all_vi_arr, float*  all_yi_arr,
                                float* all_s_arr, int* all_mask_arr,
                                thrust::host_vector<float> &H_params, thrust::host_vector<float> &H_xs, 
                                thrust::host_vector<float> &H_ys, 
                                float** H_actions,
                                thrust::host_vector<int32_t> &H_coo_len_per_ac,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS1,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS2,
                                thrust::host_vector<float>* H_Aarr_of_cooProb,
                                thrust::host_vector<float>* H_Aarr_of_Rs
                                );

void build_sparse_transition_model_at_T_at_a(int t, int action_id, int bDimx, thrust::host_vector<float> &H_tdummy, 
                                float* all_u_arr, float* all_v_arr, float* all_ui_arr,
                                float* all_vi_arr, float*  all_yi_arr,
                                float* all_s_arr, int* all_mask_arr,
                                thrust::host_vector<float> &H_params, thrust::host_vector<float> &H_xs, 
                                thrust::host_vector<float> &H_ys, 
                                float** H_actions,
                                thrust::host_vector<int32_t> &H_coo_len_per_ac,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS1,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS2,
                                thrust::host_vector<float>* H_Aarr_of_cooProb,
                                thrust::host_vector<float>* H_Aarr_of_Rs
                                ){

 
    int gsize = (int) H_params[0];
    int num_actions =  (int)H_params[1];
    int nrzns = (int) H_params[2];
    int nt = (int) H_params[10];
    int m = (int)H_params[18];
    int Nb = (m*m) + 1; //+1 is to store no. of S2s not lying in nieghbour_array. Ideally it should have 0

    float* H_T_arr = thrust::raw_pointer_cast(&H_tdummy[0]);
    float* xs = thrust::raw_pointer_cast(&H_xs[0]);
    float* ys = thrust::raw_pointer_cast(&H_ys[0]);
    float* params = thrust::raw_pointer_cast(&H_params[0]);
        
    // checks
    // if (t == nt-2){
        // std::cout << "t = " << t << "\n nt = " << nt << "\n" ; 
        // std::cout<<"check 1 : gisze= " << gsize << std::endl;
    // }
 
    // initialse master S2 array
    long long int NcNb =  ncells*Nb;
    thrust::host_vector<float> H_master_S2_vector(NcNb, 0);
    float* H_master_S2_arr = thrust::raw_pointer_cast(&H_master_S2_vector[0]);

    // initialise master sum_Rsa array - sumRsa's 
    // Important to initialise it with 0
    // need to intiilase with 0 at for each chunknum
    thrust::host_vector<float> H_master_sumRsa_vector(ncells, 0);
    float* H_master_sumRsa_arr = thrust::raw_pointer_cast(&H_master_sumRsa_vector[0]);

    float ac_speed = H_actions[action_id][0];
    float ac_angle = H_actions[action_id][1];
    // std::cout << "starting transition calc\n";
    // launch kernel for @a @t
    transition_calc(H_T_arr, 
        ncells, all_u_arr, all_v_arr, all_ui_arr, all_vi_arr, all_yi_arr,
        all_s_arr, all_mask_arr,
        ac_speed, ac_angle, xs, ys, params, H_master_sumRsa_arr, 
        H_master_S2_arr);

    // std::cout << "starting compute mean---  ncells ="<< ncells<< "\n";
    compute_mean(H_master_sumRsa_arr, ncells, nrzns);

    //initialising vectors for counting nnzs or number of uniqe S2s for S1s
    thrust::host_vector<unsigned long long int> H_num_uq_s2(ncells,0);
    thrust::host_vector<unsigned long long int> H_prSum_num_uq_s2(ncells);
    unsigned long long int* num_uq_s2_ptr = thrust::raw_pointer_cast(&H_num_uq_s2[0]);
    unsigned long long int* prSum_num_uq_s2_ptr = thrust::raw_pointer_cast(&H_prSum_num_uq_s2[0]);
    // count no. of ug unique S2 for each S1 and fill in num_uq_s2
    // std::cout << "starting count kernel-- nrzns ="<< nrzns<< "\n";
    count_kernel(H_master_S2_arr, ncells, Nb, nrzns, num_uq_s2_ptr);

    // calc nnz: number of non zero elements(or unique S2s) for a given S1 and action
    long long int nnz = thrust::reduce(H_num_uq_s2.begin(), H_num_uq_s2.end(), (float) 0, thrust::plus<float>());
    // get prefix sum of D_num_uq_s2. This helps threads to access apt COO indices in reduce_kernel
    thrust::exclusive_scan(H_num_uq_s2.begin(), H_num_uq_s2.end(), H_prSum_num_uq_s2.begin());
    // std::cout << "nnz = " << nnz<< "\n";

    //initilise coo arrays (concated across actions)
    thrust::host_vector<long long int> H_coo_s1(nnz);
    thrust::host_vector<long long int> H_coo_s2(nnz);
    thrust::host_vector<float> H_coo_count(nnz); // TODO: makde this int32_t and introduce another array for prob
    long long int* H_coo_s1_arr = thrust::raw_pointer_cast(&H_coo_s1[0]);
    long long int* H_coo_s2_arr = thrust::raw_pointer_cast(&H_coo_s2[0]);
    float* H_coo_cnt_arr = thrust::raw_pointer_cast(&H_coo_count[0]);


    reduce_kernel(H_master_S2_arr, t, Nb, m,
                    ncells, nrzns, gsize, H_coo_s1_arr, H_coo_s2_arr, H_coo_cnt_arr, 
                    num_uq_s2_ptr, prSum_num_uq_s2_ptr);

    // in algo2, this function is for one action, and I already know nnz.
    // nnz should be filled in a global array 
    H_coo_len_per_ac[action_id] = nnz;
    // Copy Device COO rusults to Host COO vectors across actions and append vectors across time
    assert(action_id >=0);
    H_Aarr_of_cooS1[action_id].insert(H_Aarr_of_cooS1[action_id].end(), H_coo_s1.begin(), H_coo_s1.end());
    H_Aarr_of_cooS2[action_id].insert(H_Aarr_of_cooS2[action_id].end(), H_coo_s2.begin(), H_coo_s2.end());
    H_Aarr_of_cooProb[action_id].insert(H_Aarr_of_cooProb[action_id].end(), H_coo_count.begin(), H_coo_count.end());
    H_Aarr_of_Rs[action_id].insert(H_Aarr_of_Rs[action_id].end(), H_master_sumRsa_vector.begin(), H_master_sumRsa_vector.end());

        //checks
        // std::cout << "H_coo_len_per_ac" << std::endl;
        // for (int n = 0; n < num_actions; n++)
        //   std::cout << H_coo_len_per_ac[n] << std::endl;

        // std::cout << "H_Aarr_of_cooS1" << std::endl;
        // for (int n = 0; n < num_actions; n++){
        //     for (int i = 0; i < H_Aarr_of_cooS1[n].size(); i++)
        //         std::cout << H_Aarr_of_cooS1[n][i] << " , " << H_Aarr_of_cooS2[n][i] << " , " << H_Aarr_of_cooProb[n][i] << std::endl;
        //     std::cout << std::endl;
        // }

        // std::cout << "H_Aarr_of_Rs" << std::endl;
        // for (int n = 0; n < num_actions; n++){
        //     for (int i = 0; i < ncells; i++)
        //         std::cout << H_Aarr_of_Rs[n][i] << std::endl;
        //     std::cout << std::endl;
        // }


        // // array of num_actions decive_vvectors for sum_Rsa_vec
        // // initialasation with 0 is important. because values are added to this
        // thrust::host_vector<float> H_arr_sumR_sa[num_actions];
        // for(int n = 0; n < num_actions; n++){
        //     H_arr_sumR_sa[n] = thrust::host_vector<float>(nnz[i]);
    

}


int get_reward_type(std::string prob_type){
    // returns 
    // 0 for time
    // 1 for energy1
    // 2 for energy2
    // 3 for energy3

    if (prob_type == "time")
        return 0;
    else if (prob_type == "energy1")
        return 1;
    else if (prob_type == "energy2")
        return 2;
    else if (prob_type == "energy3")
        return 3;
    else    
        return -1;
}


int main(int argc, char* argv[]){

// -------------------- input data starts here ---------------------------------



        // std::cout << "case_id = " << case_id << "\n";
        #include "input_to_build_model.h"
        // int num_ac_speeds = 1; //verify prob_type
        // int num_ac_angles = 16*(case_id+1);
        // int32_t num_actions = num_ac_speeds*num_ac_angles;
        nt = 5;
        gsize = strtol(argv[1], NULL, 0);


        int reward_type = get_reward_type(prob_type);
        // std::cout << "Reward type: " << reward_type << "\n";

        // define full problem name and print them to a temporary file
        // the temp file will be read by python scripts for conversion
        std::string prob_specs = get_prob_name(num_ac_speeds, num_ac_angles, i_term, 
                                                j_term, term_subgrid_size);
        std::string op_Fname_upto_prob_name = "data_modelOutput/" + prob_type + "/"
                                    + prob_name + "/" ;
        std::string op_FnamePfx = op_Fname_upto_prob_name + prob_specs + "/"; //path for storing op npy data.
        std::ofstream fout("temp_modelOp_dirName.txt");
        fout << prob_type << "\n";
        fout << prob_name << "\n";
        fout << prob_specs << "\n";
        fout << op_FnamePfx;
        fout.close();

        // TODO: 1. read paths form file xx DONE
        //       2. Make sure files are stored in np.float32 format
        std::string data_path = "data_input/" + prob_name + "/";
        std::string all_u_fname = data_path + "all_u_mat.npy";
        std::string all_v_fname = data_path + "all_v_mat.npy";
        std::string all_ui_fname = data_path + "all_ui_mat.npy";
        std::string all_vi_fname = data_path + "all_vi_mat.npy";
        std::string all_yi_fname = data_path + "all_Yi.npy";
        std::string all_s_fname = data_path + "all_s_mat.npy";
        std::string all_mask_fname = data_path + "obstacle_mask.npy"; //this file stored in int32



    // -------------------- input data ends here ---------------------------------

        // make directory for storing output data from this file
        // make_dir(op_Fname_upto_prob_name);
        // make_dir(op_FnamePfx);

        int all_u_n_elms = gsize*gsize*nt;
        int all_v_n_elms = gsize*gsize*nt;
        int all_ui_n_elms = gsize*gsize*nt*nmodes;
        int all_vi_n_elms = gsize*gsize*nt*nmodes;
        int all_yi_n_elms = nmodes*nrzns*nt;
        int all_s_n_elms = gsize*gsize*nt;
        int all_mask_n_elms = gsize*gsize*nt;

        //COMMENTING TEMPORARILY FOR LOOPED RUNS OVER PROBLEM SIZES
        //Will NOT have to create data with python scripts
        //Will just use garbage values of velocity field.

            // int all_u_n_elms;
            // int all_v_n_elms;
            // int all_ui_n_elms;
            // int all_vi_n_elms;
            // int all_yi_n_elms;
            // int all_s_n_elms;
            // int all_mask_n_elms;

            // cnpy::NpyArray all_u_cnpy = read_velocity_field_data(all_u_fname, &all_u_n_elms);
            // cnpy::NpyArray all_v_cnpy = read_velocity_field_data(all_v_fname, &all_v_n_elms);
            // cnpy::NpyArray all_ui_cnpy = read_velocity_field_data(all_ui_fname, &all_ui_n_elms);
            // cnpy::NpyArray all_vi_cnpy = read_velocity_field_data(all_vi_fname, &all_vi_n_elms);
            // cnpy::NpyArray all_yi_cnpy = read_velocity_field_data(all_yi_fname, &all_yi_n_elms);
            // cnpy::NpyArray all_s_cnpy = read_velocity_field_data(all_s_fname, &all_s_n_elms);
            // cnpy::NpyArray all_mask_cnpy = read_velocity_field_data(all_mask_fname, &all_mask_n_elms);


            // float* all_u_mat = all_u_cnpy.data<float>();
            // float* all_v_mat = all_v_cnpy.data<float>();
            // float* all_ui_mat = all_ui_cnpy.data<float>();
            // float* all_vi_mat = all_vi_cnpy.data<float>();
            // float* all_yi_mat = all_yi_cnpy.data<float>();
            // float* all_s_mat = all_s_cnpy.data<float>();
            // int* all_mask_mat = all_mask_cnpy.data<int>();
        
            // remove this once done looping over problem sizes
            float* all_u_mat = new float[all_u_n_elms];
            float* all_v_mat = new float[all_v_n_elms];
            float* all_ui_mat = new float[all_ui_n_elms];
            float* all_vi_mat = new float[all_vi_n_elms];
            float* all_yi_mat = new float[all_yi_n_elms];
            float* all_s_mat = new float[all_s_n_elms]{0};
            int* all_mask_mat = new int[all_mask_n_elms]{0};

        // print_array<float>(all_u_mat, all_u_n_elms, "all_u_mat", " ");
        // print_array<float>(all_ui_mat, all_ui_n_elms,"all_ui_mat", " ");
        // print_array<float>(all_yi_mat, all_yi_n_elms,"all_yi_mat", " ");

        // std::cout << "Finished reading Velocity Field Data !" << std::endl;
        assert(neighb_gsize <= gsize);

        //TODO: fill params in a function
        // Contains implicit casting from int32_t to float
        thrust::host_vector<float> H_params(32);
        H_params[0] = gsize;
        H_params[1] = num_actions; 
        H_params[2] = nrzns;
        H_params[3] = F;
        H_params[4] = dt;
        H_params[5] = r_outbound;
        H_params[6] = r_terminal;
        H_params[7] = nmodes;
        H_params[8] = i_term;
        H_params[9] = j_term;
        H_params[10] = nt;
        H_params[11] = is_stationary;
        H_params[12] = term_subgrid_size;
        H_params[13] = reward_type;
        H_params[14] = num_ac_speeds;
        H_params[15] = num_ac_angles;
        H_params[16] = dx;
        H_params[17] = dy;
        H_params[18] = neighb_gsize; // referred to as m in functions

        for( int i =20; i<32; i++)
            H_params[i] = z;

        // Define grid ticks in host
        thrust::host_vector<float> H_xs(gsize, -1);
        thrust::host_vector<float> H_ys(gsize, -1);
        float* xs = thrust::raw_pointer_cast(&H_xs[0]);
        float* ys = thrust::raw_pointer_cast(&H_ys[0]);
        //TODO:  2. move the fucntion to a separate file
        define_xs_or_ys(xs, dx, x0, gsize);
        define_xs_or_ys(ys, dy, y0, gsize);

        // define angles in host
        float** H_actions = new float*[num_actions];
        for(int i=0; i<num_actions; i++)
            H_actions[i] = new float[2];
        populate_actions(H_actions, num_ac_speeds, num_ac_angles, F);
        // std::cout << "CHECK:   ACTIONS:    \n";
        // for(int i=0; i<num_actions; i++){
        //     std::cout << H_actions[i][0] << ", " << H_actions[i][1] << "\n";
        // }


        // std::cout << "Copied to Device : Velocity Field Data !" << std::endl;

        thrust::host_vector<float> H_tdummy(2,0);


        // initialise reuseable host vectors
        thrust::host_vector<int32_t> H_coo_len_per_ac(num_actions);
        thrust::host_vector<long long int> H_Aarr_of_cooS1[(int)num_actions];
        thrust::host_vector<long long int> H_Aarr_of_cooS2[(int)num_actions];
        thrust::host_vector<float> H_Aarr_of_cooProb[(int)num_actions];
        thrust::host_vector<float> H_Aarr_of_Rs[(int)num_actions];
        //initialised with 0 size. later data from device is inserted/appended to the end of vector
        for (int i =0; i < num_actions; i++){
            H_Aarr_of_cooS1[i] = thrust::host_vector<long long int> (0);
        }
        for (int i =0; i < num_actions; i++){
            H_Aarr_of_cooS2[i] = thrust::host_vector<long long int> (0);
        }
        for (int i =0; i < num_actions; i++){
            H_Aarr_of_cooProb[i] = thrust::host_vector<float> (0);
        }
        for (int i =0; i < num_actions; i++){
            H_Aarr_of_Rs[i] = thrust::host_vector<float> (0);
        }

        ncells = gsize*gsize;           // assign value to global variable

        // run time loop and compute transition data for each time step
        auto start = high_resolution_clock::now(); 
        auto end = high_resolution_clock::now(); 
        auto duration_t = duration_cast<microseconds>(end - start);
        
        auto overall_start = high_resolution_clock::now(); 
        auto overall_end = high_resolution_clock::now(); 
        auto overall_duration_t = duration_cast<microseconds>(overall_end - overall_start);
        
        float first_four_ts[4];
        float total_exec_time;
        //IMP: Run time loop till nt-1. There ar no S2s to S1s in the last timestep

        for(int t = 0; t < nt-1; t++){
            // std::cout << "*** Computing data for timestep, T = " << t << std::endl;
            H_tdummy[0] = t;
            start = high_resolution_clock::now(); 
                for(int action_id = 0; action_id < num_actions; action_id++){
                    // std::cout << "  * action_id= " << action_id;
                    // this function also concats coos across time.
                    build_sparse_transition_model_at_T_at_a(t, action_id, bDimx, H_tdummy, all_u_mat, all_v_mat, 
                            all_ui_mat, all_vi_mat, all_yi_mat,
                            all_s_mat, all_mask_mat,
                            H_params, H_xs, H_ys, H_actions, 
                            H_coo_len_per_ac,
                            H_Aarr_of_cooS1, H_Aarr_of_cooS2, H_Aarr_of_cooProb,
                            H_Aarr_of_Rs);
                            //  output_data )  
                }
            end = high_resolution_clock::now(); 
            duration_t = duration_cast<microseconds>(end - start);
            if (t<4){
                first_four_ts[t] = duration_t.count()/1e6;
            }
        }

        overall_end = high_resolution_clock::now(); 
        overall_duration_t = duration_cast<microseconds>(overall_end - overall_start);
        total_exec_time = overall_duration_t.count()/1e6;

        std::cout << gsize << ",\t" << ncells << ",\t" << nt << ",\t" << ncells*nt << ",\t"  
                << num_actions << ",\t" << nrzns << ",\t" <<  nmodes << ",\t" << neighb_gsize << ",\t";
        for (int i=0; i<4; i++)
            std::cout << first_four_ts[i] << ",\t";
        std::cout << total_exec_time << "\n";
        


        // fill R vectors of each action for the last time step with high negative values. 
        // this has to be done seaprately because the above loop runs till nt-1.
        /*
            TODO: 1. Verify rewards as last time step
        */
        thrust::host_vector<float> H_rewards_at_end_t(ncells, 0);
        for (int i =0; i < num_actions; i++){
            H_Aarr_of_Rs[i].insert(H_Aarr_of_Rs[i].end(), H_rewards_at_end_t.begin(), H_rewards_at_end_t.end());
        }
        //Check
        // for (int i =0; i < num_actions; i++)
        //     std::cout << H_Aarr_of_Rs[i].size() << " ";
        

        // find nnz per action
        thrust::host_vector<long long int> H_master_PrSum_nnz_per_ac(num_actions);
        long long int DP_relv_params[2] = {ncells*nt, num_actions*1LL};

        long long int master_nnz = 0;       //running sum of nnz going across actions
        // calculate inclusive prefix sum of nnz's across actions 
        // will be used to access indeces while concatenating results across across actions
        for(int i = 0; i < num_actions; i++){
            master_nnz += H_Aarr_of_cooS1[i].size();
            H_master_PrSum_nnz_per_ac[i] = master_nnz;
        }

        // print_array<long long int>(DP_relv_params, 2, "DP_relv_params", " ");
        unsigned long int num_DP_params = sizeof(DP_relv_params) / sizeof(DP_relv_params[0]);
        // std::cout << "chek num = " << sizeof(DP_relv_params) << std::endl;
        // std::cout << "chek denom = " << sizeof(DP_relv_params[0]) << std::endl;

        // //checks
        // std::cout << "total/master_nnz = " << master_nnz << std::endl;
        // std::cout << "H_Aarr_of_cooS1[i].size()" << std::endl;
        // for(int i = 0; i < num_actions; i++)
        //     std::cout << H_Aarr_of_cooS1[i].size() << std::endl;
        // print_array<long long int>(&H_Aarr_of_cooS2[0][0], 10,  "H_Aarr_of_cooS2[0]", " ");


        // save final coo data
        thrust::host_vector<long long int> H_master_cooS1(master_nnz);
        thrust::host_vector<long long int> H_master_cooS2(master_nnz);
        thrust::host_vector<float> H_master_cooVal(master_nnz);
        thrust::host_vector<float> H_master_R(ncells*nt*num_actions, -99999); //TODO: veriffy -99999
        // save_master_Coos_to_file(op_FnamePfx, num_actions,
        //                             H_master_cooS1, 
        //                             H_master_cooS2, 
        //                             H_master_cooVal,
        //                             H_master_R,
        //                             H_Aarr_of_cooS1,
        //                             H_Aarr_of_cooS2,
        //                             H_Aarr_of_cooProb,
        //                             H_Aarr_of_Rs,
        //                             H_params,
        //                             DP_relv_params,
        //                             num_DP_params);

    
    return 0;
}

//------------ main ends here ------------------------------------------


void save_master_Coos_to_file(std::string op_FnamePfx, int num_actions,
    thrust::host_vector<long long int> &H_master_cooS1, 
    thrust::host_vector<long long int> &H_master_cooS2, 
    thrust::host_vector<float> &H_master_cooVal,
    thrust::host_vector<float> &H_master_R,
    thrust::host_vector<long long int>* H_Aarr_of_cooS1,
    thrust::host_vector<long long int>* H_Aarr_of_cooS2,
    thrust::host_vector<float>* H_Aarr_of_cooProb,
    thrust::host_vector<float>* H_Aarr_of_Rs,
    thrust::host_vector<float> &prob_params,
    long long int* DP_relv_params,
    unsigned long int num_DP_params){
    //  Convertes floats to int32 for COO row and col idxs
    //  copies from each action vector to a master vector
    //  master_coo vectors is concatation first across time, then across action
    //  ALSO, MODIFIES S1(t,i,j) to S1(t,i,j,a)

    unsigned long long int master_nnz = H_master_cooS1.size();
    unsigned long long int prob_params_size = prob_params.size();
    long long int m_idx = 0;
    int n_states = DP_relv_params[0];

    std::cout << "in save \n" ;

    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooS1[i].size(); j++){
            // TODO: modify to include actions
            H_master_cooS1[m_idx] = H_Aarr_of_cooS1[i][j] + i*n_states;
            m_idx++;
        }
    }

    std::cout << "in save \n" ;
    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooS2[i].size(); j++){
            H_master_cooS2[m_idx] = H_Aarr_of_cooS2[i][j];
            m_idx++;
        }
    }

    std::cout << "in save \n" ;
    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooProb[i].size(); j++){
            H_master_cooVal[m_idx] = H_Aarr_of_cooProb[i][j];
            m_idx++;
        }
    }

    std::cout << "in save \n" ;
    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_Rs[i].size(); j++){
            H_master_R[m_idx] = H_Aarr_of_Rs[i][j];
            m_idx++;
        }
    }

    //checks
    // std::cout << "check num_DP_params = " << num_DP_params << std::endl;
    // std::cout << "op_FnamePfx= " <<  op_FnamePfx << "\n";
    
    cnpy::npy_save(op_FnamePfx + "master_cooS1.npy", &H_master_cooS1[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_cooS2.npy", &H_master_cooS2[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_cooVal.npy", &H_master_cooVal[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_R.npy", &H_master_R[0], {H_master_R.size(),1},"w");
    cnpy::npy_save(op_FnamePfx + "DP_relv_params.npy", &DP_relv_params[0], {num_DP_params,1},"w");
    cnpy::npy_save(op_FnamePfx + "prob_params.npy", &prob_params[0], {prob_params_size,1},"w");

}



cnpy::NpyArray read_velocity_field_data( std::string file_path_name, int* n_elements){
    // reads numpy file from input and 
    // returns cnpy::NpyArray stucture  and also fills in num_elements in the passed reference n_elements
    // extraction in main: float* vel_data = arr.data<float>();
    // TODO: make it general. currently hard-coded for float arrays.

    //print filename
    std::cout << "file path and name:   " << file_path_name << std::endl;
    cnpy::NpyArray arr = cnpy::npy_load(file_path_name);

    //prints for checks 
    int dim = arr.shape.size();
    int num_elements = 1;
    std::cout << "shape: " ;
    for (int i = 0; i < dim; i++){
        std::cout << arr.shape[i] << " , " ;
        num_elements = num_elements*arr.shape[i];
    }
    *n_elements = num_elements;
    std::cout << std::endl << "num_elements: " << num_elements << std::endl;

    float* vel_data = arr.data<float>();
    // print check first 10 elements
    std::cout << "First 10 elements of loaded array are: " << std::endl;
    for (int i = 0; i < 10; i++)
         std::cout << vel_data[i] << "  " ;
    
    std::cout << std::endl << std::endl;

    return arr;

}



void make_dir(std::string dir_name){
    int mkdir_status;
    std::string comm_mkdir = "mkdir ";
    std::string str = comm_mkdir + dir_name;
    const char * full_command = str.c_str();
    mkdir_status = system(full_command);
    std::cout << "mkdir_status = " << mkdir_status << std::endl;
}



void define_xs_or_ys(float* xs, float dx, float x0, int gsize){

    for(int i = 0; i < gsize;  i++)
        xs[i] = x0 + i*dx;
}



void populate_ac_angles(float* ac_angles, int num_ac_angles){
    //fills array with equally spaced angles in radians
    for (int i = 0; i < num_ac_angles; i++)
        ac_angles[i] = i*(2*M_PI)/num_ac_angles;
    return;
}



void populate_ac_speeds(float* ac_speeds, int num_ac_speeds, float Fmax){
    //fills array with ac_speeds
    // std::cout << "infunc CHeck- num_ac_speeds = " << num_ac_speeds << "\n";
    float delF = 0;
    if (num_ac_speeds == 1)
        ac_speeds[0] = Fmax;
    else if (num_ac_speeds > 1){
        // -----include 0 speed
        // delF = Fmax/(num_ac_speeds-1);
        // for(int i = 0; i<num_ac_speeds; i++)
        //     ac_speeds[i] = i*delF;
        // ------exclude 0 speed
        delF = Fmax/(num_ac_speeds);
        for(int i = 0; i<num_ac_speeds; i++){
            ac_speeds[i] = (i+1)*delF;
            std::cout << ac_speeds[i] << "\n";
        }
    }
    else
        std::cout << "Invalid num_ac_speeds\n";
    
    return;
}


void populate_actions(float **H_actions, int num_ac_speeds, int num_ac_angles, float Fmax){
    // populates 2d vector with possible actions
    float* ac_angles = new float[num_ac_angles];
    populate_ac_angles(ac_angles, num_ac_angles);

    float* ac_speeds = new float[num_ac_speeds];
    populate_ac_speeds(ac_speeds, num_ac_speeds, Fmax);

    int idx;
    for (int i=0; i<num_ac_speeds; i++){
        for(int j=0; j<num_ac_angles; j++){
            idx = j + num_ac_angles*i;
            // std::cout << ac_speeds[i] << "\n";
            H_actions[idx][0] = ac_speeds[i];
            H_actions[idx][1] = ac_angles[j];
        }
    }

    return;
}